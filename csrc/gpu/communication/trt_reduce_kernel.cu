// reference: https://github.com/NVIDIA/TensorRT-LLM/blob/release/0.14/cpp/tensorrt_llm/kernels/customAllReduceKernels.h

// #include <c10/cuda/CUDAStream.h>

#include <cassert>
#include <iostream>
#include <sstream>
#include <unordered_map>
#include <sys/mman.h>
#include <sys/wait.h>
#include <unistd.h>

#include "trt_reduce_internal.cuh"

using namespace trt_llm;

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << "\n";
        exit(EXIT_FAILURE);
    }
}

using fptr_t = int64_t;

class AllReduceMeta {
 public:
  AllReduceMeta(int64_t rank_id, int64_t world_size, const std::vector<fptr_t>& buffers,
                const std::vector<fptr_t>& barrier_in, const std::vector<fptr_t>& barrier_out) {
    this->rank_id = (int)rank_id;
    this->world_size = (int)world_size;
    this->buffers = buffers;
    this->barrier_in = barrier_in;
    this->barrier_out = barrier_out;
  }

 public:
  int world_size;
  int rank_id;
  std::vector<fptr_t> buffers;
  std::vector<fptr_t> barrier_in;
  std::vector<fptr_t> barrier_out;
  int barrier_flag = 1;

  static std::shared_ptr<AllReduceMeta> instance;
};

std::shared_ptr<AllReduceMeta> AllReduceMeta::instance = nullptr;

#define NUM_PROCESSES 8
struct SharedMemory {
    fptr_t buffer_array[NUM_PROCESSES];
    fptr_t barrier_in_array[NUM_PROCESSES];
    fptr_t barrier_out_array[NUM_PROCESSES];

    SharedMemory() {
        for (int i = 0; i < NUM_PROCESSES; ++i) {
            buffer_array[i] = -1;
            barrier_in_array[i] = -1;
            barrier_out_array[i] = -1;
        }
    }

    void barrier(int world_size){
        for(;;){
        bool reduce_is_done = true;
        for(int i=0; i< world_size; ++i){
          if(this->buffer_array[i] == -1 || this->barrier_in_array[i] == -1 || this->barrier_out_array[i] == -1){
            reduce_is_done = false;
          }
        }
        if(reduce_is_done){
          break;
        }
        usleep(1); // sleep 1ms
      }
    }
};

// Get the number of bits for a given data type.
inline int get_bits(paddle::DataType dtype) {
  switch (dtype) {
    case paddle::DataType::FLOAT32:
      return 32;
    case paddle::DataType::BFLOAT16:
    case paddle::DataType::FLOAT16:
      return 16;
    default:
      assert(false && "Unsupported data type");
  }
}

// Check if customized all-reduce kernels can be applied.
inline bool CanApplyCustomAllReduce(int64_t num_elements, paddle::DataType dtype) {
  // The customized all-reduce kernel has the following requirement(s).
  return num_elements % (16 / ((get_bits(dtype) + 7) / 8)) == 0;
}

void init_custom_ar(int64_t rank_id, int64_t world_size) {
  
  const int buffer_max_size = 8 * 1024 * 1024;
  const int barrier_max_size = 8 * (24 + 2) * 8;
  static std::mutex init_mutex;
  std::lock_guard<std::mutex> lock(init_mutex);
  if (!AllReduceMeta::instance) {
    void* shared_memory = mmap(nullptr, sizeof(SharedMemory),
                               PROT_READ | PROT_WRITE,
                               MAP_SHARED | MAP_ANONYMOUS, -1, 0);
    SharedMemory* shm = static_cast<SharedMemory*>(shared_memory);
    if (shared_memory == MAP_FAILED) {
        perror("mmap");
        return;
    }
    
    void* buffers_ptr = nullptr;
    checkCudaError(hipMalloc(&buffers_ptr, buffer_max_size), "hipMalloc ptr error");
    void* barrier_in_ptr = nullptr; 
    checkCudaError(hipMalloc(&barrier_in_ptr, barrier_max_size), "hipMalloc ptr error");
    void* barrier_out_ptr = nullptr;
    checkCudaError(hipMalloc(&barrier_out_ptr, barrier_max_size), "hipMalloc ptr error");
    hipIpcMemHandle_t buffers_hanlder;
    checkCudaError(hipIpcGetMemHandle(&buffers_hanlder, buffers_ptr), "Failed to get buffers IPC memory handle");
    hipIpcMemHandle_t barrier_in_hanlder;
    checkCudaError(hipIpcGetMemHandle(&barrier_in_hanlder, barrier_in_ptr), "Failed to get barrier_in IPC memory handle");
    hipIpcMemHandle_t barrier_out_hanlder;
    checkCudaError(hipIpcGetMemHandle(&barrier_out_hanlder, barrier_out_ptr), "Failed to get barrier_out IPC memory handle");

    shm->buffer_array[rank_id] = (fptr_t)&buffers_hanlder;
    shm->barrier_in_array[rank_id] = (fptr_t)&barrier_in_hanlder;
    shm->barrier_out_array[rank_id] = (fptr_t)&barrier_out_hanlder;

    std::vector<fptr_t> buffers, barrier_in, barrier_out;
    for(int i=0; i< world_size; ++i){
      buffers.emplace_back(shm->buffer_array[i]);
      barrier_in.emplace_back(shm->barrier_in_array[i]);
      barrier_out.emplace_back(shm->barrier_out_array[i]);
    }

    shm->barrier(int(world_size));

    AllReduceMeta::instance = std::make_shared<AllReduceMeta>(rank_id, world_size, buffers, barrier_in, barrier_out);

    munmap(shared_memory, sizeof(SharedMemory));
  }
}

void dispose(fptr_t _fa) {
  auto fa = reinterpret_cast<AllReduceMeta*>(_fa);
  delete fa;
}

void all_reduce(const paddle::Tensor& inp, paddle::Tensor& out, int rank_id, int world_size) {
  if (!AllReduceMeta::instance) {
    init_custom_ar(rank_id, world_size);
  }
  auto m = AllReduceMeta::instance;
  auto stream = inp.stream();
  auto num_elements = inp.numel();
  auto dtype = inp.type();
  AllReduceStrategyType strategy = SelectImplementation(num_elements * ((get_bits(dtype) + 7) / 8), m->world_size);

  // should be gurantee in python code
  assert(strategy == AllReduceStrategyType::ONESHOT || strategy == AllReduceStrategyType::TWOSHOT);
  assert(CanApplyCustomAllReduce(num_elements, dtype));

  // Initialize the all-reduce kernel arguments.

  AllReduceParams params;
  params.ranks_per_node = world_size;
  params.rank = rank_id;
  params.local_rank = rank_id;
  params.local_input_buffer_ptr = const_cast<void *>(inp.data());
  params.local_output_buffer_ptr = out.data();
  params.elts_total = inp.numel();
  params.elts_size = 2; // TODO for bfloat16 and float16
  params.barrier_flag = ++(m->barrier_flag);

  for (int i = 0; i < world_size; ++i) {
    params.peer_comm_buffer_ptrs[i] = reinterpret_cast<void*>(m->buffers[i]);
  }
  for (int i = 0; i < world_size; ++i) {
    params.peer_barrier_ptrs_in[i] = reinterpret_cast<uint32_t*>(m->barrier_in[i]);
  }
  for (int i = 0; i < world_size; ++i) {
    params.peer_barrier_ptrs_out[i] = reinterpret_cast<uint32_t*>(m->barrier_out[i]);
  }

  auto data_type = out.type();
  trtCustomAllReduce(params, data_type, strategy, stream);
}

std::vector<std::vector<int64_t>> TrtReduceInferShape(
    const std::vector<int64_t>& inp_shape) {
  return {inp_shape};
}

std::vector<paddle::DataType> TrtReduceInferDtype(
    const paddle::DataType& inp_dtype) {
  return {inp_dtype};
}

PD_BUILD_OP(trt_reduce)
    .Inputs({"inp"})
    .Outputs({"output"})
    .Attrs({"rank_id: int", "world_size: int"})
    .SetKernelFn(PD_KERNEL(all_reduce))
    .SetInferShapeFn(PD_INFER_SHAPE(TrtReduceInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(TrtReduceInferDtype));